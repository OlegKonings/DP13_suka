#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <fstream>
#include <sstream>
#include <utility>
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <string>
#include <cmath>
//#include <map>
#include <ctime>
#include <hip/hip_runtime.h>
#include <math_functions.h>
#include "hip/hip_runtime.h"
#include ""
#include <Windows.h>
#include <MMSystem.h>
#pragma comment(lib, "winmm.lib")
#define _CRTDBG_MAP_ALLOC
#include <crtdbg.h>
using namespace std;

#define _DTH hipMemcpyDeviceToHost
#define _DTD hipMemcpyDeviceToDevice
#define _HTD hipMemcpyHostToDevice
#define THREADS_SMALL 64
#define THREADS_LARGE 256

bool InitMMTimer(UINT wTimerRes);
void DestroyMMTimer(UINT wTimerRes, bool init);
inline int choose2(int n){return n>0 ? ((n*(n-1))>>1):0;}

double CPU_version(double *DP0, double *DP1, const int N){
	double ret=0.;
	const unsigned int adj=N+2;
	memset(DP0,0,adj*adj*sizeof(double));
	DP0[adj]=1.;

	for(int i=0;i<N;i++){//steps taken forward towards N
		if(i&1)memset(DP0,0,adj*adj*sizeof(double));
		else
			memset(DP1,0,adj*adj*sizeof(double));
		for(int j=1;j<=(i+1);j++){//some length amount which could have been reached this turn(length)
			double t;
			for(int k=0;k<j;k++){//k represents the current postions
				if(i&1){
					t=DP1[j*adj+k];//where may have been 1 step back,j index is current length, k is current location
					if(t>0.){//was reached before
						t*=0.5;
						if(0==k)DP0[(j+1)*adj]+=t;//fill in new boundray for next iter
						else
							DP0[j*adj+(k-1)]+=t;//to left
						if((j-1)==k)DP0[(j+1)*adj+j]+=t;//fill in new boundray for next iter
						else
							DP0[j*adj+(k+1)]+=t;//to right
					}
				}else{
					t=DP0[j*adj+k];
					if(t>0.){
						t*=0.5;
						if(0==k)DP1[(j+1)*adj]+=t;
						else
							DP1[j*adj+(k-1)]+=t;
						if((j-1)==k)DP1[(j+1)*adj+j]+=t;
						else
							DP1[j*adj+(k+1)]+=t;
					}
				}
			}
		}
	}
	for(int i=1;i<=(N+1);i++){
		for(int j=0;j<i;j++){
			if(!(N&1))ret+=double(i)*DP0[i*adj+j];
			else
				ret+=double(i)*DP1[i*adj+j];
		}
	}

	return ret;
}

__device__ double atomicAdd(double* address, double val){
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do{
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    }while(assumed != old);
    return __longlong_as_double(old);
}

__device__ __forceinline__ int d_choose2(int n){return n>0 ? ((n*(n-1))>>1):0;}
__device__ __forceinline__ int d_num_combo(int a, int b){return a>b ? (d_choose2(a)+b):(d_choose2(b)+a);}

__device__  __forceinline__ double shfl_d64(double x,int lane){
	return __hiloint2double( __shfl( __double2hiint(x), lane ), __shfl( __double2loint(x), lane ));
}

__global__ void GPU_step0(const int i,const double* __restrict__ DP_prev, double* __restrict__ DP_cur, const int adj,const int N){
	const int j=threadIdx.x+blockIdx.x*blockDim.x;
	const int k=blockIdx.y;
	if(j>(i+1) || (k>=j) )return;
	double t= DP_prev[j*adj+k];
	if(t>0.){
		t*=0.5;
		if(0==k)atomicAdd(&DP_cur[(j+1)*adj],t);
		else
			atomicAdd(&DP_cur[j*adj+(k-1)],t);
		if((j-1)==k)atomicAdd(&DP_cur[(j+1)*adj+j],t);
		else
			atomicAdd(&DP_cur[j*adj+(k+1)],t);
	}
}

__global__ void GPU_step1(const double* __restrict__ DP,double* __restrict__ D_ans, const int adj, const int N,const int bound){//will assume DP pointer is correct from host
	const int offset = blockIdx.x*blockDim.x + threadIdx.x;
	const int warp_index=threadIdx.x%32;

	__shared__ double b_val[8];
	double t_val=0.0f;

	if(offset<bound){
		int lo=0,hi=adj,cur,mid,j=offset,i;
		while(lo<hi){
			mid=(hi+lo+1)>>1;
			cur=d_choose2(mid);
			if(cur>j)hi=mid-1;
			else
				lo=mid;
		}
		j-=d_choose2(lo);
		i=lo;
		if(i>0 && (i<(N+2)) && (j<i) ){
			t_val=((double)(i)*DP[i*adj+j]);
		}
	}
	t_val+=shfl_d64(t_val,warp_index+16);
	t_val+=shfl_d64(t_val,warp_index+8);
	t_val+=shfl_d64(t_val,warp_index+4);
	t_val+=shfl_d64(t_val,warp_index+2);
	t_val+=shfl_d64(t_val,warp_index+1);
	if(warp_index==0){
		b_val[threadIdx.x>>5]=t_val;
	}
	__syncthreads();
	if(threadIdx.x==0){
		atomicAdd(&D_ans[0],(b_val[0]+b_val[1]+b_val[2]+b_val[3]+b_val[4]+b_val[5]+b_val[6]+b_val[7]));
	}
}


int main(){
	const unsigned int num_spaces=1000;
	cout<<"\nnum= "<<num_spaces<<'\n';
	const unsigned int problem_space=(num_spaces+2)*(num_spaces+2);
	const unsigned int num_bytes=problem_space*sizeof(double);
	double *DP0=(double *)malloc(num_bytes);
	double *DP1=(double *)malloc(num_bytes);
	double CPU_ans=0.,GPU_ans=0.;

	hipError_t err=hipDeviceReset();
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

	UINT wTimerRes = 0;
	DWORD CPU_time=0,GPU_time=0;
    bool init = InitMMTimer(wTimerRes);
    DWORD startTime=timeGetTime();

	CPU_ans=CPU_version(DP0,DP1,num_spaces);

	DWORD endTime = timeGetTime();
    CPU_time=endTime-startTime;

    cout<<"CPU solution timing: "<<CPU_time<<'\n';
	cout<<"CPU answer= "<<CPU_ans<<'\n';

	int ii=0;
	const int adj=num_spaces+2,N=num_spaces;
	const int bound=choose2(adj);
	const double s_val=1.;

	double *D_DP0,*D_DP1,*D_ans;
	err=hipMalloc((void**)&D_DP0, num_bytes);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	err=hipMalloc((void**)&D_DP1, num_bytes);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	err=hipMalloc((void**)&D_ans, sizeof(double));
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

	dim3 grid0(1,1,1);

	wTimerRes = 0;
	init = InitMMTimer(wTimerRes);
	startTime = timeGetTime();

	err=hipMemset(D_DP0,0,num_bytes);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	err=hipMemset(D_ans,0,sizeof(double));
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	err=hipMemcpy(D_DP0+adj,&s_val,sizeof(double),_HTD);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	
	for(;ii<N;ii++){
		grid0.x=( ((ii+2)+THREADS_SMALL-1)/THREADS_SMALL);
		grid0.y=(ii+1);

		if(ii&1){
			hipMemset(D_DP0,0,num_bytes);
			GPU_step0<<<grid0,THREADS_SMALL>>>(ii,D_DP1,D_DP0,adj,N);

		}else{
			hipMemset(D_DP1,0,num_bytes);
			GPU_step0<<<grid0,THREADS_SMALL>>>(ii,D_DP0,D_DP1,adj,N);
		}
		err=hipDeviceSynchronize();
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	}
	grid0.x=(bound+THREADS_LARGE-1)/THREADS_LARGE;
	grid0.y=1;
	if(N&1){
		GPU_step1<<<grid0,THREADS_LARGE>>>(D_DP1,D_ans,adj,N,bound);
	}else{
		GPU_step1<<<grid0,THREADS_LARGE>>>(D_DP0,D_ans,adj,N,bound);
	}
	err=hipDeviceSynchronize();
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

	err=hipMemcpy(&GPU_ans,D_ans,sizeof(double),_DTH);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

	endTime = timeGetTime();
	GPU_time=endTime-startTime;
	cout<<"\nCUDA timing(including all memory transfers and ops): "<<GPU_time<<" , answer= "<<GPU_ans<<'\n';
	DestroyMMTimer(wTimerRes, init);


	err=hipFree(D_DP0);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	err=hipFree(D_DP1);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	err=hipFree(D_ans);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	
	free(DP0);
	free(DP1);
	return 0;
}

bool InitMMTimer(UINT wTimerRes){
	TIMECAPS tc;
	if (timeGetDevCaps(&tc, sizeof(TIMECAPS)) != TIMERR_NOERROR) {return false;}
	wTimerRes = min(max(tc.wPeriodMin, 1), tc.wPeriodMax);
	timeBeginPeriod(wTimerRes); 
	return true;
}

void DestroyMMTimer(UINT wTimerRes, bool init){
	if(init)
		timeEndPeriod(wTimerRes);
}


